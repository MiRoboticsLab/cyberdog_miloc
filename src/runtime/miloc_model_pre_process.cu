#include "hip/hip_runtime.h"
// Copyright (c) 2023 Beijing Xiaomi Mobile Software Co., Ltd. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "runtime/miloc_model_pre_process.hpp"

namespace cyberdog
{
namespace miloc
{

static int GetBlocks(const int N)
{
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

__global__ void local_pre_norm_kernel(const uint8_t *__restrict src, float *dst, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= size)
        return;
    if (index >= 240 * 640){
        dst[index] = 1.0;
    }else{
        dst[index] = (float) src[index] / 255.f;
    }
}

__global__ void global_pre_norm_kernel(const uint8_t *__restrict src, float *dst, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= size)
        return;
    dst[index] = (float) src[index] / 255.f;
}

int LocalModelAPreProcess(MilocMat &src, MilocMat &dst)
{
    int size = src.shape_[1] * src.shape_[2];
    
    local_pre_norm_kernel<<<GetBlocks(size), CUDA_NUM_THREADS>>>(
        (uint8_t *)src.data_, (float *)dst.data_, size);
    
    return hipGetLastError();
}

int GlobalModelAPreProcess(MilocMat &src, MilocMat &dst)
{
    int size = src.shape_[1] * src.shape_[2];
    
    global_pre_norm_kernel<<<GetBlocks(size), CUDA_NUM_THREADS>>>(
        (uint8_t *)src.data_, (float *)dst.data_, size);
    
    return hipGetLastError();;
}

}  // namespace miloc
}  // namespace cyberdog